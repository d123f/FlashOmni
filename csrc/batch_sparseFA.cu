/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cstdint>
#include <sparseqkv/attention/mask.cuh>
#include <sparseqkv/attention/scheduler.cuh>
#include <sparseqkv/pos_enc.cuh>
#include <optional>
#include <iostream>

#include "batch_sparseFA_config.inc"
#include "pytorch_conversion_utils.h"
#include "pytorch_extension_utils.h"

namespace sparseqkv {

template <uint32_t CTA_TILE_Q, uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO,
          PosEncodingMode POS_ENCODING_MODE, bool USE_FP16_QK_REDUCTION, MaskMode MASK_MODE,
          typename AttentionVariant, typename Params>
hipError_t BatchSparseFAWithRaggedKVDispatched(Params params, typename Params::DTypeO* tmp_v,
                                                    float* tmp_s, hipStream_t stream);

}  // namespace sparseqkv

using namespace sparseqkv;

at::Tensor BatchSparseFAWithKVPlan(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor qo_indptr, at::Tensor kv_indptr,
    at::Tensor kv_len_arr, int64_t total_num_rows, int64_t batch_size, int64_t num_qo_heads,
    int64_t num_kv_heads, int64_t page_size, bool enable_cuda_graph, int64_t head_dim_qk,
    int64_t head_dim_vo) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * float_workspace_buffer.element_size();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * int_workspace_buffer.element_size();

  PrefillPlanInfo plan_info;

  const c10::cuda::OptionalCUDAGuard device_guard(float_workspace_buffer.device());
  const hipStream_t stream = c10::cuda::getCurrentCUDAStream();
  hipError_t status = PrefillPlan<IdType>(
      float_workspace_buffer.data_ptr(), float_workspace_size_in_bytes,
      int_workspace_buffer.data_ptr(), page_locked_int_workspace_buffer.data_ptr(),
      int_workspace_size_in_bytes, plan_info, qo_indptr.data_ptr<IdType>(),
      kv_indptr.data_ptr<IdType>(), total_num_rows, batch_size, num_qo_heads, num_kv_heads,
      head_dim_qk, head_dim_vo, page_size, enable_cuda_graph, /*sizeof_dtype_o=*/2, stream);

  TORCH_CHECK(status == hipSuccess,
              "Failed to plan prefill with error: ", hipGetErrorString(status));

  return vec_to_tensor(plan_info.ToVector());
}

void BatchSparseFAWithRaggedKVRun(at::Tensor float_workspace_buffer,
                                      at::Tensor int_workspace_buffer, at::Tensor plan_info_vec,
                                      at::Tensor q, at::Tensor k, at::Tensor v, 
                                      std::optional<at::Tensor> sparse_info, std::optional<at::Tensor> sparse_kv_info,
                                      std::optional<at::Tensor> sparse_info_indptr, std::optional<at::Tensor> sparse_kv_info_indptr, 
                                      int64_t sparse_block_size_for_q, int64_t sparse_block_size_for_kv, bool is_full,
                                      at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor o,
                                      std::optional<at::Tensor> maybe_lse, int64_t mask_mode_code,
                                      int64_t layout ADDITIONAL_FUNC_PARAMS) {
  PrefillPlanInfo plan_info;
  plan_info.FromVector(tensor_to_vec(plan_info_vec));
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);

  int64_t num_qo_heads = q.size(1);
  int64_t head_dim_qk = q.size(2);
  int64_t num_kv_heads = (kv_layout == QKVLayout::kNHD) ? k.size(1) : k.size(0);
  // q: (B * N, H, D)
  uint32_t q_stride_n = q.stride(0), q_stride_h = q.stride(1), k_stride_n, k_stride_h, v_stride_n,
           v_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    k_stride_n = k.stride(0);
    k_stride_h = k.stride(1);
    v_stride_n = v.stride(0);
    v_stride_h = v.stride(1);
  } else {
    k_stride_h = k.stride(0);
    k_stride_n = k.stride(1);
    v_stride_h = v.stride(0);
    v_stride_n = v.stride(1);
  }

  if (maybe_lse) {
    const auto& lse = *maybe_lse;
    TORCH_CHECK(lse.size(0) == q.size(0), lse.size(0), q.size(0));
    TORCH_CHECK(lse.size(1) == q.size(1), lse.size(1), q.size(1));
  }

  void* float_buffer_ptr = float_workspace_buffer.data_ptr();
  void* int_buffer_ptr = int_workspace_buffer.data_ptr();

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k.scalar_type();

  const c10::cuda::OptionalCUDAGuard device_guard(float_workspace_buffer.device());
  const hipStream_t stream = c10::cuda::getCurrentCUDAStream();
  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
      USE_LOGITS_SOFT_CAP, USE_FP16_QK_REDUCTION, AttentionVariant,
      RaggedSparseFAParams, [&] {
        RaggedSparseFAParams params;
        params.q = static_cast<DTypeQ*>(q.data_ptr());
        params.k = static_cast<DTypeKV*>(k.data_ptr());
        params.v = static_cast<DTypeKV*>(v.data_ptr());
        params.o = static_cast<DTypeO*>(o.data_ptr());
        params.sparse_info = sparse_info.has_value() ? static_cast<uint8_t*>(sparse_info->data_ptr()) : nullptr;
        params.sparse_kv_info = sparse_kv_info.has_value() ? static_cast<uint8_t*>(sparse_kv_info->data_ptr()) : nullptr;
        params.sparse_info_indptr =
            sparse_info_indptr.has_value() ? static_cast<IdType*>(sparse_info_indptr->data_ptr()) : nullptr;
        params.sparse_kv_info_indptr =
            sparse_kv_info_indptr.has_value() ? static_cast<IdType*>(sparse_kv_info_indptr->data_ptr()) : nullptr;
        params.sparse_block_size_for_q = sparse_block_size_for_q;
        params.sparse_block_size_for_kv = sparse_block_size_for_kv;
        params.is_full = is_full;
        params.lse = maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
        params.q_indptr = static_cast<IdType*>(qo_indptr.data_ptr());
        params.kv_indptr = static_cast<IdType*>(kv_indptr.data_ptr());
        params.num_qo_heads = num_qo_heads;
        params.num_kv_heads = num_kv_heads;
        params.group_size = uint_fastdiv(num_qo_heads / num_kv_heads);
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.k_stride_n = k_stride_n;
        params.k_stride_h = k_stride_h;
        params.v_stride_n = v_stride_n;
        params.v_stride_h = v_stride_h;

        params.request_indices = nullptr;
        params.qo_tile_indices = nullptr;
        params.kv_tile_indices = nullptr;
        params.merge_indptr = nullptr;
        params.o_indptr = nullptr;
        params.kv_chunk_size_ptr = nullptr;
        params.block_valid_mask = nullptr;
        params.total_num_rows = nullptr;
        params.max_total_num_rows = 0;
        params.padded_batch_size = 0;
        params.partition_kv = false;

        ADDITIONAL_PARAMS_SETTER

        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;

        params.request_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.request_indices_offset);
        params.qo_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
        params.kv_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_tile_indices_offset);
        params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.o_indptr_offset);
        params.kv_chunk_size_ptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_chunk_size_ptr_offset);
        if (plan_info.split_kv) {
          params.merge_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
          tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.v_offset);
          tmp_s = GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.s_offset);
          if (plan_info.enable_cuda_graph) {
            params.block_valid_mask =
                GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
          }
        }
        params.padded_batch_size = plan_info.padded_batch_size;
        params.max_total_num_rows = plan_info.total_num_rows;
        if (plan_info.enable_cuda_graph) {
          params.total_num_rows =
              GetPtrFromBaseOffset<uint32_t>(int_buffer_ptr, plan_info.total_num_rows_offset);
        }

        hipError_t status = hipSuccess;

        DISPATCH_CTA_TILE_Q(plan_info.cta_tile_q, CTA_TILE_Q, {
          status = sparseqkv::BatchSparseFAWithRaggedKVDispatched<
              CTA_TILE_Q, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
              /*use_fp16_qk_reduction=*/USE_FP16_QK_REDUCTION, MASK_MODE, AttentionVariant,
              RaggedSparseFAParams>(params, tmp_v, tmp_s, stream);
        });

        TORCH_CHECK(status == hipSuccess, "BatchSparseFAWithRaggedKV failed with error ",
                    hipGetErrorString(status));
        return true;
      });
}
